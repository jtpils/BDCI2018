/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
 * The modifications are
 *      removed texture memory usage
 *      removed split query KNN computation
 *      added feature extraction with bilinear interpolation
 *
 * Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
 */

// Includes
#include <cstdio>
#include "hip/hip_runtime.h"

// Constants used by the program
#define BLOCK_DIM                      16


//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//
__global__ void extract_with_interpolation(
    int nthreads,
    float *data, float *n_xy_coords, float *extracted_data,
    int n_max_coord, int channels, int height, int width) {

  int x0, x1, y0, y1, nc;
  float wx0, wx1, wy0, wy1;
  int n, nd;
  float x, y;

  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < (nthreads);
       index += blockDim.x * gridDim.x) {
    n = (index / n_max_coord);
    nd = n * n_max_coord * channels;
    x = n_xy_coords[index * 2];
    y = n_xy_coords[index * 2 + 1];

    x0 = static_cast<int>(floor(x));
    x1 = x0 + 1;
    y0 = static_cast<int>(floor(y));
    y1 = y0 + 1;

    x0 = x0 <= 0 ? 0 : (x0 >= (width - 1)  ? (width - 1) : x0);
    y0 = y0 <= 0 ? 0 : (y0 >= (height - 1) ? (height - 1) : y0);
    x1 = x1 <= 0 ? 0 : (x1 >= (width - 1)  ? (width - 1) : x1);
    y1 = y1 <= 0 ? 0 : (y1 >= (height - 1) ? (height - 1) : y1);

    wx0 = static_cast<float>(x1) - x;
    wx1 = x - x0;
    wy0 = static_cast<float>(y1) - y;
    wy1 = y - y0;

    if(x0 == x1){ wx0 = 1; wx1 = 0; }
    if(y0 == y1){ wy0 = 1; wy1 = 0; }
    for(int c=0; c < channels; c++) {
      nc = (n * channels + c) * height;
      // extracted_data[index * channels + c] = wy0 * wx0 * data[(nc + y0) * width + x0]
      // extracted_data[nd + index % n_max_coord + n_max_coord * c] = index;
      extracted_data[nd + index % n_max_coord + n_max_coord * c] = wy0 * wx0 * data[(nc + y0) * width + x0]
       + wy1 * wx0 * data[(nc + y1) * width + x0]
       + wy0 * wx1 * data[(nc + y0) * width + x1]
       + wy1 * wx1 * data[(nc + y1) * width + x1];
    }
  }
}

/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceGlobal(const float* A, int wA,
    const float* B, int wB, int dim,  float* AB){

  // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A  = BLOCK_DIM * wA;
  step_B  = BLOCK_DIM * wB;
  end_A   = begin_A + (dim-1) * wA;

    // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
  int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
    if (a/wA + ty < dim){
      shared_A[ty][tx] = (cond0)? A[a + wA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1)? B[b + wB * ty + tx] : 0;
    }
    else{
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
    if (cond2 && cond1){
      for (int k = 0; k < BLOCK_DIM; ++k){
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp*tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[(begin_A + ty) * wB + begin_B + tx] = ssd;
}


/**
  * Gathers k-th smallest distances for each column of the distance matrix in the top.
  *
  * @param dist        distance matrix
  * @param ind         index matrix
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, int *ind, int width, int height, int k){

  // Variables
  int l, i, j;
  float *p_dist;
  int   *p_ind;
  float curr_dist, max_dist;
  int   curr_row,  max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex<width){
    // Pointer shift, initialization, and max value
    p_dist   = dist + xIndex;
    p_ind    = ind  + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 1;

    // Part 1 : sort kth firt elementZ
    for (l=1; l<k; l++){
      curr_row  = l * width;
      curr_dist = p_dist[curr_row];
      if (curr_dist<max_dist){
        i=l-1;
        for (int a=0; a<l-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=l; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
      } else {
        p_ind[l*width] = l+1;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k-1)*width;
    for (l=k; l<height; l++){
      curr_dist = p_dist[l*width];
      if (curr_dist<max_dist){
        i=k-1;
        for (int a=0; a<k-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=k-1; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
        max_dist             = p_dist[max_row];
      }
    }
  }
}


/**
  * Computes the square root of the first line (width-th first element)
  * of the distance matrix.
  *
  * @param dist    distance matrix
  * @param width   width of the distance matrix
  * @param k       number of neighbors to consider
  */
__global__ void cuParallelSqrt(float *dist, int width, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (xIndex<width && yIndex<k)
    dist[yIndex*width + xIndex] = sqrt(dist[yIndex*width + xIndex]);
}


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//


/**
  * Prints the error message return during the memory allocation.
  *
  * @param error        error value return by the memory allocation function
  * @param memorySize   size of memory tried to be allocated
  */
void printErrorMessage(hipError_t error, int memorySize){
  printf("==================================================\n");
  printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
  printf("Whished allocated memory : %d\n", memorySize);
  printf("==================================================\n");
}

void myknnLauncher(int batch_size, int qrs_num, int pts_num, int channels_num,
                 const float *queries, const float *points, int k,
                 float *out_dis, int *out_ids){
    for(int batch_id=0; batch_id<batch_size; batch_id++) {
        int query_width = qrs_num;
        int ref_width = pts_num;
        const float *ref_dev = points + pts_num * channels_num * batch_size;
        const float *query_dev = queries + qrs_num * channels_num * batch_size;
        int height = channels_num;
        float *dist_dev = out_dis + qrs_num * pts_num * batch_size;
        int *ind_dev = out_ids + qrs_num * k * 2 * batch_size;

        // Grids ans threads
        dim3 g_16x16(query_width/16, ref_width/16, 1);
        dim3 t_16x16(16, 16, 1);
        if (query_width%16 != 0) g_16x16.x += 1;
        if (ref_width  %16 != 0) g_16x16.y += 1;
        //
        dim3 g_256x1(query_width/256, 1, 1);
        dim3 t_256x1(256, 1, 1);
        if (query_width%256 != 0) g_256x1.x += 1;

        dim3 g_k_16x16(query_width/16, k/16, 1);
        dim3 t_k_16x16(16, 16, 1);
        if (query_width%16 != 0) g_k_16x16.x += 1;
        if (k  %16 != 0) g_k_16x16.y += 1;

        // Kernel 1: Compute all the distances
        cuComputeDistanceGlobal<<<g_16x16,t_16x16>>>(ref_dev, ref_width,
            query_dev, query_width, height, dist_dev);

        // Kernel 2: Sort each column
        cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, ind_dev,
            query_width, ref_width, k);

        // Kernel 3: Compute square root of k first elements
        cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, k);
    }
}